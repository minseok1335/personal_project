
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int InputData[5] = {1,2,3,4,5};
    int OutputData[5] = {0};

    int* GraphicsCard_Mem;

    hipMalloc ((void**)&GraphicsCard_Mem, 5 * sizeof(int));
    hipMemcpy (GraphicsCard_Mem, InputData, 5 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy (OutputData, GraphicsCard_Mem, 5 * sizeof(int), hipMemcpyDeviceToHost);

    for (int i=0; i<5; i++) {
        printf("Output[%d]: %d\n", i, OutputData[i]);
    }

    hipFree (GraphicsCard_Mem);
    return 0;
}