#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void vector_add (int *a, int *b, int *c) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	c[idx] = a[idx] * b[idx];

}

int main() {

	int i = 0;
	int nBlocks = 1024;
	int nThreads = 512;
	int size = nBlocks * nThreads;
	size_t BufferSize = size * sizeof(int);

	int *h_a, *h_b, *h_c;
	int *d_a, *d_b, *d_c;

	hipHostAlloc ((void**)&h_a, BufferSize, hipHostMallocMapped);
	hipHostAlloc ((void**)&h_b, BufferSize, hipHostMallocMapped);
	hipHostAlloc ((void**)&h_c, BufferSize, hipHostMallocMapped);

	for (i = 0; i < size; i++) {
		h_a[i] = i;
		h_b[i] = i;
	}

	hipHostGetDevicePointer ((void**) &d_a, (void*)h_a, 0);
	hipHostGetDevicePointer ((void**) &d_b, (void*)h_b, 0);
	hipHostGetDevicePointer ((void**) &d_c, (void*)h_c, 0);

	vector_add <<<nBlocks, nThreads>>> (d_a, d_b, d_c);
	hipDeviceSynchronize();

	hipHostFree(h_a);
	hipHostFree(h_b);
	hipHostFree(h_c);

	return 0;

}
