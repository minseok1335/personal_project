
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


int main() {

	const int size = 1024 * 1024 * 200;
	const int BufferSize = size * sizeof(int);

	int *page_able_memory_in;
	int *page_able_memory_out;

	page_able_memory_in = (int*)malloc(BufferSize);
	page_able_memory_out = (int*)malloc(BufferSize);

	for (int i = 0; i < size; i++) {
		page_able_memory_in[i] = i;
		page_able_memory_out[i] = 0;
	}

	int *DeviceMemory;

	hipMalloc ((void**)&DeviceMemory, BufferSize);

	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	hipMemcpy(DeviceMemory, page_able_memory_in, BufferSize, hipMemcpyHostToDevice);
	hipMemcpy(page_able_memory_out, DeviceMemory, BufferSize, hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsed_time;

	hipEventElapsedTime (&elapsed_time, start, stop);

	printf("transfer time: %lf msec\n",elapsed_time);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	free(page_able_memory_in);
	free(page_able_memory_out);

	
	int *pinned_memory_in;
	int *pinned_memory_out;

	hipHostMalloc ((void**)&pinned_memory_in, BufferSize, hipHostMallocDefault);
	hipHostMalloc ((void**)&pinned_memory_out, BufferSize, hipHostMallocDefault);

	for (int i = 0; i < size; i++) {
		pinned_memory_in[i] = i;
		pinned_memory_out[i] = 0;
	}

	
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);


	hipMemcpy(DeviceMemory, pinned_memory_in, BufferSize, hipMemcpyHostToDevice);
	hipMemcpy(pinned_memory_out, DeviceMemory, BufferSize, hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime (&elapsed_time, start, stop);
	printf("transfer time(pinned memory): %lf msec\n",elapsed_time);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipHostFree(pinned_memory_in);
	hipHostFree(pinned_memory_out);

	hipFree(DeviceMemory);

	return 0;
}




