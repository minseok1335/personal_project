
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel (int *in, int *out) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = 0; i < 5; i++)
		out[tid] += in[tid];
}

int main () {

	const int nStreams = 15;
	const int nBlocks = 65535;
	const int nThreads = 512;
	const int N = 512 * 65535;
	const int size = N * sizeof(int);


	int *h_in;
	int *h_out;

	hipHostMalloc ((void**)&h_in, size, hipHostMallocDefault);
	hipHostMalloc ((void**)&h_out, size, hipHostMallocDefault);

	for (int i = 0; i < N; i++) {
		h_in[i] = i;
		h_out[i] = 0;
	}

	int *d_in;
	int *d_out;

	hipMalloc ((void**)&d_in, size);
	hipMalloc ((void**)&d_out, size);

	hipMemset(d_in, 0, size);
	hipMemset(d_out, 0, size);

	hipEvent_t SyncStart, SyncStop;

	float SyncTime;

	hipEventCreate(&SyncStart);
	hipEventCreate(&SyncStop);

	hipEventRecord(SyncStart, 0);

	hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

	kernel <<<nBlocks, nThreads>>> (d_in, d_out);

	hipMemcpy(d_out, h_out, size, hipMemcpyDeviceToHost);

	hipEventRecord(SyncStop, 0);
	hipEventSynchronize(SyncStop);

	hipEventElapsedTime(&SyncTime, SyncStart, SyncStop);

	printf("synchronization version: %f msec\n", SyncTime);

	for (int i = 0; i < N; i++) {
		h_in[i] = i;
		h_out[i] = 0;
	}

	hipMemset(d_in, 0, size);
	hipMemset(d_out, 0, size);

	hipStream_t *streams = (hipStream_t*)malloc(nStreams * sizeof(hipStream_t));

	for (int i = 0; i < nStreams; i++) 
		hipStreamCreate(&(streams[i]));
	
	hipEvent_t StreamStart, StreamStop;
	float StreamTime;

	hipEventCreate(&StreamStart);
	hipEventCreate(&StreamStop);

	int offset = 0;
	int chunck_size = size / nStreams;

	hipEventRecord(StreamStart, 0);

	for (int i = 0; i < nStreams; i++) {
		offset = i * N / nStreams;
		hipMemcpyAsync(d_in + offset, h_in + offset, chunck_size, hipMemcpyHostToDevice, streams[i]);
	}

	for (int i = 0; i < nStreams; i++) {
		offset = i * N / nStreams;
		kernel <<<nBlocks/nStreams, nThreads, 0, streams[i]>>> (d_in + offset, d_out + offset);
	}

	for (int i = 0; i < nStreams; i++) {
		offset = i * N / nStreams;
		hipMemcpyAsync(h_in + offset, d_out + offset, chunck_size, hipMemcpyDeviceToHost, streams[i]);
	}

	hipEventRecord(StreamStop, 0);
	hipEventSynchronize(StreamStop);

	hipEventElapsedTime (&StreamTime, StreamStart, StreamStop);

	printf("stream version: %f\n", StreamTime);

	hipEventDestroy(SyncStart);
	hipEventDestroy(SyncStop);
	hipEventDestroy(StreamStart);
	hipEventDestroy(StreamStop);

	for (int i = 0; i < nStreams; i++)
		hipStreamDestroy(streams[i]);

	hipFree (d_in);
	hipFree (d_out);

	hipHostFree (h_in);
	hipHostFree (h_out);

	return 0;
}
