
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void CountAtomic (int *count) {
	atomicAdd(count, 1);
}

__global__ void CountShared (int *count) {

	__shared__ int nCount;
	
	if(threadIdx.x == 0)
		nCount = 0;
	__syncthreads();

	atomicAdd(&nCount, 1);
	__syncthreads();

	if (threadIdx.x == 0)
		atomicAdd (count, nCount);
}

int main() {
	const int nBlocks = 10000;
	const int nThreads = 512;

	int host_nThreadCount = 0;
	int *dev_nThreadCount;

	hipMalloc((void**)&dev_nThreadCount, sizeof(int));
	hipMemset(dev_nThreadCount, 0, sizeof(int));

	hipEvent_t start, stop;
	float Elapsed_time;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	CountAtomic <<<nBlocks, nThreads>>> (dev_nThreadCount);

	hipMemcpy(&host_nThreadCount, dev_nThreadCount, sizeof(int), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&Elapsed_time, start, stop);

	printf("global memory\n threads: %d\n elapsed time: %f\n", host_nThreadCount, Elapsed_time);

	hipEventDestroy(start);
	hipEventDestroy(stop);


	host_nThreadCount = 0;
	hipMemset(dev_nThreadCount, 0, sizeof(int));

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	CountShared <<<nBlocks, nThreads>>> (dev_nThreadCount);

	hipMemcpy(&host_nThreadCount, dev_nThreadCount, sizeof(int), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&Elapsed_time, start, stop);

	printf("shared memory\n threads: %d\n elapsed time: %f\n", host_nThreadCount, Elapsed_time);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(dev_nThreadCount);
	return 0;
}
