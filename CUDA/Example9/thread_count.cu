
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void ThreadRace (int *count) {
	(*count)++;
}

__global__ void ThreadAtomic (int *count) {
	atomicAdd(count, 1);
}

int main () {
	const int nBlocks = 10000;
	const int nThreads = 512;

	int count = 0;
	int *dev_count;

	hipMalloc ((void**)&dev_count, sizeof(int));
	hipMemset (dev_count, 0, sizeof(int));

	ThreadRace <<<nBlocks, nThreads>>> (dev_count);

	hipMemcpy (&count, dev_count, sizeof(int), hipMemcpyDeviceToHost);

	printf("race count: %d\n", count);


	count = 0;
	hipMemset (dev_count, 0, sizeof(int));

	ThreadAtomic <<<nBlocks, nThreads>>> (dev_count);

	hipMemcpy (&count, dev_count, sizeof(int), hipMemcpyDeviceToHost);

	printf("atomic count: %d\n", count);

	hipFree(dev_count);
	return 0;
}
